#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "kittens.cuh"
using namespace kittens;

#define NUM_THREADS (kittens::WARP_THREADS)

#define _row 16
#define _col 32
struct fr_globals {
    using _gl = gl<float, -1, -1, -1, -1, st_fl<_row, _col>>;
    _gl x;
    float accum;
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void fr_tk(const __grid_constant__ fr_globals g) {
    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int *)&__shm[0]);
    st_fl<_row, _col> (&x_s) = al.allocate<st_fl<_row, _col>>();

    rt_fl<_row, _col> x_reg;

    rt_fl<1, 1> accum_reg;
    zero(accum_reg);



    load(x_s, g.x, {0, 0, 0, 0});
    __syncthreads();

    load(x_reg, x_s);
    __syncthreads();

    mul(x_reg, x_reg, x_reg);
    __syncthreads();

    sum(accum_reg, x_reg);
    __syncthreads();

    load(g.accum, accum_reg);
}

void dispatch_fr_tk(float *x, float *accum) {
    using _gl = gl<float, -1, -1, -1, -1, st_fl<_row, _col>>;
    using globals = fr_globals;
    _gl x_gl{x, 1, 1, _row, _col};
    globals g{x_gl, 0.0f};
    fr_tk<<<1, 32>>>(g);
    hipDeviceSynchronize();
}